#include "hip/hip_runtime.h"
	#include <hip/hip_runtime.h>
	#include <omp.h>
	#include <stdlib.h>
	#include <math.h>
	#include <array>
	#include <vector>
	#include <sstream>
	#include <chrono>
	#include <hiprand/hiprand_kernel.h>
	#include <limits>
	#include <memory>
	#include <cstddef>
	#include <type_traits>

	#include "../include/timer.cuh"
	#include "../include/musket.cuh"
	#include "../include/spfb64_0.cuh"
	
	
	
	const double PI = 3.141592653589793;
	
	//Float2::Float2() : x(), y() {}
	

	
	struct FIR_map_index_in_place_array_functor{
		
		FIR_map_index_in_place_array_functor(const mkt::DArray<float>& _input, const mkt::DArray<float>& _coeff) : input(_input), coeff(_coeff){}
		
		~FIR_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int Index, float2 a){
			float2 newa;
			newa.x = 0.0f;
			newa.y = 0.0f;
			
			if(((Index) <= ((channels) * (spectra)))){
			for(int j = 0; ((j) < (taps)); j++){
				newa.x += (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				input.get_data_local(((Index) + ((j) * (channels))))
				 * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				coeff.get_data_local(((Index%(taps*channels)) + ((j) * (channels))))
				);
			}
			}
			return (newa);
		}
	
		void init(int device){
			input.init(device);
			coeff.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		int taps;
		int channels;
		int spectra;
		
		mkt::DeviceArray<float> input;
		mkt::DeviceArray<float> coeff;
	};
	struct Fetch_map_index_in_place_array_functor{
		
		Fetch_map_index_in_place_array_functor(const mkt::DArray<float2>& _c_output) : c_output(_c_output){}
		
		~Fetch_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int i, float2 Ti){
			return // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			c_output.get_data_local((i ^ (int) __powf(2, (((log2size) - 1) - (counter)))))
			;
		}
	
		void init(int device){
			c_output.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		int counter;
		int log2size;
		
		mkt::DeviceArray<float2> c_output;
	};
	struct Combine_map_index_in_place_array_functor{
		
		Combine_map_index_in_place_array_functor(const mkt::DArray<float2>& _c_input_double) : c_input_double(_c_input_double){}
		
		~Combine_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int Index, float2 Ai){
			float2 newa;
			newa.x = 0.0f;
			newa.y = 0.0f;
			int b = Index >> (log2size - counter - 1);
			int b2 = 0;

			for(int l = 0;l <= counter;l++) {
				b2 = (b & 1) ? 2 * b2 + 1 : 2 * b2;
				b >>= 1;
			}

			double temp = 2.0 * pi / Problemsize * (b2 << (log2size - counter - 1));
			float2 intermediateresult;
			intermediateresult.x = __cosf(temp);
			intermediateresult.y = -__sinf(temp);
			
			if(((Index) == __powf(2, (((log2size) - 1) - (counter))))){
			float2 mult_res;
			mult_res.x = (((intermediateresult).x * (Ai).x) - ((intermediateresult).y * (Ai).y));
			mult_res.y = (((intermediateresult).x * (Ai).y) + ((intermediateresult).y * (Ai).x));
			float2 add_res;
			add_res.x = (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			c_input_double.get_data_local((Index)).x
			 + (mult_res).x);
			add_res.y = (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			c_input_double.get_data_local((Index)).y
			 + (mult_res).y);
			newa = (add_res);
			}
			 else {
					float2 mult_res2;
					mult_res2.x = (((intermediateresult).x * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).x
					) - ((intermediateresult).y * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).y
					));
					mult_res2.y = (((intermediateresult).x * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).y
					) + ((intermediateresult).y * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).x
					));
					float2 add_res2;
					add_res2.x = ((Ai).x + (mult_res2).x);
					add_res2.y = ((Ai).y + (mult_res2).y);
					newa = (add_res2);
				}
			return (newa);
		}
	
		void init(int device){
			c_input_double.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		int counter;
		int log2size;
		double pi;
		int Problemsize;
		
		mkt::DeviceArray<float2> c_input_double;
	};
	
	
	
	
	
	
	
	int main(int argc, char** argv) {
		mkt::init();
		
		
		mkt::sync_streams();
		std::chrono::high_resolution_clock::time_point complete_timer_start = std::chrono::high_resolution_clock::now();
		GpuTimer timer;
		double allocation = 0.0,fill = 0.0, rest = 0.0, rest2 = 0.0, out = 0.0;
		timer.Start();
		mkt::DArray<float> input(0, 134218224, 134218224, 0.0f, 1, 0, 0, mkt::DIST, mkt::COPY);
		//mkt::DArray<float> input_double(0, 134217728, 134217728, 0.0f, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float2> c_input_double(0, 134217728, 134217728, float2{}, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float2> c_output(0, 134217728, 134217728, float2{}, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float> coeff(0, 1024, 1024, 0.0f, 1, 0, 0, mkt::DIST, mkt::COPY);
		timer.Stop();
		allocation += timer.Elapsed();
		// timer.Start();
		srand(1);
		for (int n = 0; n < 134218224; n++) {
			input[n] = (rand() / (float)RAND_MAX);
		}
		for (int n = 0; n < 1024; n++) {
			coeff[n] = (rand() / (float)RAND_MAX);
		}
		timer.Start();
		input.update_devices();
		coeff.update_devices();
		timer.Stop();
		fill += timer.Elapsed();
		timer.Start();
		FIR_map_index_in_place_array_functor fIR_map_index_in_place_array_functor{input, coeff};
		//Float_to_float2_map_index_in_place_array_functor float_to_float2_map_index_in_place_array_functor{input_double};
		Fetch_map_index_in_place_array_functor fetch_map_index_in_place_array_functor{c_output};
		Combine_map_index_in_place_array_functor combine_map_index_in_place_array_functor{c_input_double};
		timer.Stop();
		rest += timer.Elapsed();


		double fir_time = 0.0, fft_time = 0.0, R2C_time = 0.0;

		int ntaps = 16;
		int nchans = 64;
		int nspectra = 2097152;
		int log2size = 6;
		timer.Start();
		fIR_map_index_in_place_array_functor.taps = (ntaps);fIR_map_index_in_place_array_functor.channels = (nchans);fIR_map_index_in_place_array_functor.spectra = (nspectra);
		mkt::map_index_in_place<float2, FIR_map_index_in_place_array_functor>(c_input_double, fIR_map_index_in_place_array_functor);
		timer.Stop();
		fir_time += timer.Elapsed();
		timer.Start();
		//mkt::map_index_in_place<float2, Float_to_float2_map_index_in_place_array_functor>(c_output, float_to_float2_map_index_in_place_array_functor);
		timer.Stop();
		R2C_time += timer.Elapsed();
		timer.Start();
		for(int j = 0; ((j) < (log2size)); j++){
			fetch_map_index_in_place_array_functor.counter = (j);fetch_map_index_in_place_array_functor.log2size = (log2size);
			mkt::map_index_in_place<float2, Fetch_map_index_in_place_array_functor>(c_input_double, fetch_map_index_in_place_array_functor);
			combine_map_index_in_place_array_functor.counter = (j);combine_map_index_in_place_array_functor.log2size = (log2size);combine_map_index_in_place_array_functor.pi = (PI);combine_map_index_in_place_array_functor.Problemsize = 16;
			mkt::map_index_in_place<float2, Combine_map_index_in_place_array_functor>(c_output, combine_map_index_in_place_array_functor);
		}
		
		mkt::sync_streams();
		timer.Stop();
		fft_time += timer.Elapsed();
		timer.Start();
                c_output.update_self();
                timer.Stop();
		out += timer.Elapsed();
		printf("\n%f;%f;%f;%f;%f;%f;%f\n", fir_time, fft_time, R2C_time, allocation, fill, rest, out);
		return EXIT_SUCCESS;
		}
