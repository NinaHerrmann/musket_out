#include "hip/hip_runtime.h"
	#include <hip/hip_runtime.h>
	#include <omp.h>
	#include <stdlib.h>
	#include <math.h>
	#include <array>
	#include <vector>
	#include <sstream>
	#include <chrono>
	#include <hiprand/hiprand_kernel.h>
	#include <limits>
	#include <memory>
	#include <cstddef>
	#include <type_traits>

	#include "../include/timer.cuh"
	#include "../include/filehelper.h"
	#include "../include/musket.cuh"
	#include "../include/spfb64_0.cuh"
	
	
	
	const double PI = 3.141592653589793;
	
	//Float2::Float2() : x(), y() {}
	

	
	struct FIR_map_index_in_place_array_functor{
		
		FIR_map_index_in_place_array_functor(const mkt::DArray<float>& _input, const mkt::DArray<float>& _coeff) : input(_input), coeff(_coeff){}
		
		~FIR_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int Index, float a){
			float newa = 0;
			
			if(((Index) <= ((channels) * (spectra)))){
			for(int j = 0; ((j) < (taps)); j++){
				newa += (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				input.get_data_local(((Index) + ((j) * (channels))))
				 * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				coeff.get_data_local(((Index) + ((j) * (channels))))
				);
			}
			}
			return (newa);
		}
	
		void init(int device){
			input.init(device);
			coeff.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		int taps;
		int channels;
		int spectra;
		
		mkt::DeviceArray<float> input;
		mkt::DeviceArray<float> coeff;
	};
	struct Float_to_float2_map_index_in_place_array_functor{
		
		Float_to_float2_map_index_in_place_array_functor(const mkt::DArray<float>& _input_double) : input_double(_input_double){}
		
		~Float_to_float2_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int x, float2 y){
			y.x = static_cast<float>(// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			input_double.get_data_local((x))
			);
			y.y = 0.0f;
			return (y);
		}
	
		void init(int device){
			input_double.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		
		mkt::DeviceArray<float> input_double;
	};
	struct Fetch_map_index_in_place_array_functor{
		
		Fetch_map_index_in_place_array_functor(const mkt::DArray<float2>& _c_output) : c_output(_c_output){}
		
		~Fetch_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int i, float2 Ti){
			return // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			c_output.get_data_local((i ^ (int) __powf(2, (((log2size) - 1) - (counter)))))
			;
		}
	
		void init(int device){
			c_output.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		int counter;
		int log2size;
		
		mkt::DeviceArray<float2> c_output;
	};
	struct Combine_map_index_in_place_array_functor{
		
		Combine_map_index_in_place_array_functor(const mkt::DArray<float2>& _c_input_double) : c_input_double(_c_input_double){}
		
		~Combine_map_index_in_place_array_functor() {}
		
		__device__
		auto operator()(int Index, float2 Ai){
			float2 newa;
			newa.x = 0.0f;
			newa.y = 0.0f;
			int b = Index >> (log2size - counter - 1);
			int b2 = 0;

			for(int l = 0;l <= counter;l++) {
				b2 = (b & 1) ? 2 * b2 + 1 : 2 * b2;
				b >>= 1;
			}

			double temp = 2.0 * pi / n * (b2 << (log2size - counter - 1));
			float2 intermediateresult;
			intermediateresult.x = __cosf(temp);
			intermediateresult.y = __sinf(temp);
			
			if(((Index) == __powf(2, (((log2size) - 1) - (counter))))){
			float2 mult_res;
			mult_res.x = (((intermediateresult).x * (Ai).x) - ((intermediateresult).y * (Ai).y));
			mult_res.y = (((intermediateresult).x * (Ai).y) + ((intermediateresult).y * (Ai).x));
			float2 add_res;
			add_res.x = (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			c_input_double.get_data_local((Index)).x
			 + (mult_res).x);
			add_res.y = (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
			c_input_double.get_data_local((Index)).y
			 + (mult_res).y);
			newa = (add_res);
			}
			 else {
					float2 mult_res2;
					mult_res2.x = (((intermediateresult).x * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).x
					) - ((intermediateresult).y * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).y
					));
					mult_res2.y = (((intermediateresult).x * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).y
					) + ((intermediateresult).y * // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
					c_input_double.get_data_local((Index)).x
					));
					float2 add_res2;
					add_res2.x = ((Ai).x + (mult_res2).x);
					add_res2.y = ((Ai).y + (mult_res2).y);
					newa = (add_res2);
				}
			return (newa);
		}
	
		void init(int device){
			c_input_double.init(device);
		}
		
		size_t get_smem_bytes(){
			size_t result = 0;
			return result;
		}
		
		int counter;
		int log2size;
		double pi;
		int Problemsize;
		
		mkt::DeviceArray<float2> c_input_double;
	};
	
	
	
	
	
	
	
	int main(int argc, char** argv) {
		mkt::init();
		
		
		mkt::sync_streams();
		std::chrono::high_resolution_clock::time_point complete_timer_start = std::chrono::high_resolution_clock::now();
		
		mkt::DArray<float> input(0, 134218224, 134218224, 0.0f, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float> input_double(0, 134217728, 134217728, 0.0f, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float2> c_input_double(0, 134217728, 134217728, float2{}, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float2> c_output(0, 134217728, 134217728, float2{}, 1, 0, 0, mkt::DIST, mkt::COPY);
		mkt::DArray<float> coeff(0, 1024, 1024, 0.0f, 1, 0, 0, mkt::DIST, mkt::COPY);
		srand(1);
		for (int n = 0; n < 134316032; n++) {
			input[n] = (rand() / (float)RAND_MAX);
		}
		for (int n = 0; n < 131072; n++) {
			coeff[n] = (rand() / (float)RAND_MAX);
		}
		input.update_devices();
		coeff.update_devices();
		FIR_map_index_in_place_array_functor fIR_map_index_in_place_array_functor{input, coeff};
		Float_to_float2_map_index_in_place_array_functor float_to_float2_map_index_in_place_array_functor{input_double};
		Fetch_map_index_in_place_array_functor fetch_map_index_in_place_array_functor{c_output};
		Combine_map_index_in_place_array_functor combine_map_index_in_place_array_functor{c_input_double};
		
		
		GpuTimer timer;
		double fir_time = 0.0, fft_time = 0.0, R2C_time = 0.0;

		int ntaps = 16;
		int nchans = 64;
		int nspectra = 2097152;
		int log2size = 6;
		timer.Start();
		fIR_map_index_in_place_array_functor.taps = (ntaps);fIR_map_index_in_place_array_functor.channels = (nchans);fIR_map_index_in_place_array_functor.spectra = (nspectra);
		mkt::map_index_in_place<float, FIR_map_index_in_place_array_functor>(input_double, fIR_map_index_in_place_array_functor);
		timer.Stop();
		fir_time += timer.Elapsed();
		timer.Start();
		mkt::map_index_in_place<float2, Float_to_float2_map_index_in_place_array_functor>(c_output, float_to_float2_map_index_in_place_array_functor);
		timer.Stop();
		R2C_time += timer.Elapsed();
		timer.Start();
		for(int j = 0; ((j) < (log2size)); j++){
			fetch_map_index_in_place_array_functor.counter = (j);fetch_map_index_in_place_array_functor.log2size = (log2size);
			mkt::map_index_in_place<float2, Fetch_map_index_in_place_array_functor>(c_input_double, fetch_map_index_in_place_array_functor);
			combine_map_index_in_place_array_functor.counter = (j);combine_map_index_in_place_array_functor.log2size = (log2size);combine_map_index_in_place_array_functor.pi = (PI);combine_map_index_in_place_array_functor.Problemsize = 16;
			mkt::map_index_in_place<float2, Combine_map_index_in_place_array_functor>(c_output, combine_map_index_in_place_array_functor);
		}
		
		mkt::sync_streams();
		timer.Stop();
		fft_time += timer.Elapsed();
		printf("%.5f;%.5f;%.5f\n", fir_time, fft_time, R2C_time);
		return EXIT_SUCCESS;
		}
